#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : matrixMult.cu
 Author      : Julian Fietkau
 Copyright   : Julian Fietkau

 Implementation of matrix multiplication with CUDA
 BUILD & TESTED ONLY ON OSX 10.9 MAVERICKS with NVIDIA GTX 760 (Hackintosh)
 --> Install nvcc
  * Download NVIDA DEV TOOLKIT
  * export nvcc filepath:
      export PATH=/Developer/NVIDIA/CUDA-5.5/bin:$PATH
      export DYLD_LIBRARY_PATH=/Developer/NVIDIA/CUDA-5.5/lib:$DYLD_LIBRARY_PATH
  * after cd, running 'make run clean' on terminal
 ============================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "main.h"
#include "help.h"
#include "test.h"

// max. Threadsblockgroeße, depends on your GPU 
const int BLOCK_SIZE = 32; 

// Kernel function, multiplication, ! GPU CODE
__global__ void matMulKernel(const Matrix a, const Matrix b, Matrix c)
{
  float cvalue = 0.0f; // Zwischensumme
  // Threadblock * 32 + Subthread im Threadblock
  int row = (blockIdx.y * blockDim.y) + threadIdx.y; // Zeilenindex
  int col = (blockIdx.x * blockDim.x) + threadIdx.x; // Spaltenindex
 
  // only calulate if current thread is in place of C
  // waste some power, but is very easy solution for max BLOCK_SIZE problems
  if((row < c.rows) && (col < c.columns)){
    for (int e = 0; e < a.columns; ++e){
      cvalue += a.values[(row * a.columns) + e] * b.values[(e * b.columns) + col];
    }
    c.values[(row * c.columns) + col] = cvalue;
  }
}

// prepare GPU for computaion
void runMultiplication(const Matrix &m1, const Matrix &m2, Matrix &m3){
  
  int size;
  
  // TODO Check if multiplication is possible
  createMatrix(&m3, m1.rows, m2.columns);

  Matrix A; // represents matrix copy on device
    A.rows = m1.rows;
    A.columns = m1.columns;
    // calc size of matrix 1
    size = m1.rows * m1.columns * sizeof(float);
    // allocate memory for values
    hipMalloc((void**) &A.values, size);
    // copy given data into memory
    hipMemcpy( A.values, m1.values, size, hipMemcpyHostToDevice);

  Matrix B; // represents matrix copy on device
    B.rows = m2.rows;
    B.columns = m2.columns;
    // calc size of matrix 2
    size = m2.rows * m2.columns * sizeof(float);
    hipMalloc((void**) &B.values, size); // allocate mem for values
    // copy given data into memory
    hipMemcpy( B.values, m2.values, size, hipMemcpyHostToDevice);
     
  Matrix C; // Device-Matrix C (ErgebnisMatrix)
    C.rows = m1.rows;
    C.columns = m2.columns;
    // calc size of result matrix
    size = m3.rows * m3.columns * sizeof(float);
    // allocate mem for result values
    hipMalloc((void**) &C.values, size);
  
  // can only multiply multiples from BLOCK_SIZE, depends on your GPU
  // dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE);

  // amount of threadblocks we need (Zeilen der Matrix / verfuegbare Zeilen pro Threadblock)
  // zB. 100 x 100 Matrix --> 100 / 32 = 4 Threadbloecke noetig
  // +1 to round up every time ( 5 / 32 = 0, but we need 1 )
  dim3 dimGrid( (m3.rows / dimBlock.x) +1, (m3.columns / dimBlock.y) +1);

  // printf("%d / %d -> GridSpalten = %d\n", m3.rows, dimBlock.x, dimGrid.x );

  // ohne Shared Memory, Kernel starten
  matMulKernel<<<dimGrid, dimBlock>>>(A, B, C);
    
  // copy calculated result from device to host
  hipMemcpy( m3.values, C.values, size, hipMemcpyDeviceToHost);

  // free allocated mem
  hipFree(A.values);
  hipFree(B.values);
  hipFree(C.values);
}

int main()  {
  test();

  Matrix Matrix1;
  Matrix Matrix2;
  Matrix Matrix3;
  Matrix Matrix4;

  createIdentityMatrix(&Matrix1, 500);
  bench(Matrix1);
  destroyMatrix(&Matrix1);

  createIdentityMatrix(&Matrix2, 750);
  bench(Matrix2);
  destroyMatrix(&Matrix2);


  createIdentityMatrix(&Matrix3, 2000);
  bench(Matrix3);
  destroyMatrix(&Matrix3);


  // 10.000 * 10.000 * 4 ... / 1024 / 1024 => 380 MB 
  createIdentityMatrix(&Matrix4, 10000);
  bench(Matrix4);
  destroyMatrix(&Matrix4);
  
  // root( (8GB * 1024^3) / 4 ) = 46340
  // Testsystem with 8 GB RAM, matrix > 46340 can't allocate 
  // ! wrong computation i guess, but no error
  // Matrix Matrix5;
  // createIdentityMatrix(&Matrix5, 46340); 
  // bench(Matrix5);
  // destroyMatrix(&Matrix5);


}

